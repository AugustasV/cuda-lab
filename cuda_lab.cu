#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ unsigned int count = 0;
__shared__ bool SyncAllThreads;
//
// Only been capable to calculate matrix using cuda, no idea how to implement subset problem
//
__global__ void sum(const float* array, unsigned int N,float* result)
{
    // Each block sums a subset of the input array
    float partialSum = 0;

    if (threadIdx.x == 0) {
        // Thread 0 of each block stores the partial sum
        // to global memory
        result[blockIdx.x] = partialSum;

        // Thread 0 makes sure its result is visible to
        // all other threads
        __threadfence();

        // Thread 0 of each block signals that it is done
        unsigned int value = atomicInc(&count, gridDim.x);

        // Thread 0 of each block determines if its block is
        // the last block to be done
        SyncAllThreads = (value == (gridDim.x - 1));
    }

    // Synchronize to make sure that each thread reads
    // the correct value of SyncAllThreads
// must be reached by all threads from block
    __syncthreads();

    if (SyncAllThreads) 
    {
        // The last block sums the partial sums
        // stored in result[0 .. gridDim.x-1]
        float totalSum =  0;

        if (threadIdx.x == 0)
        {
            // Thread 0 of last block stores total sum
            // to global memory and resets count so that
            // next kernel call works properly
            result[0] = totalSum;
            count = 0;
        }
    }
}

// add all elemnts to the matrix, MatrixSumAll iterates all rows and columns in each matrix, adding A and B and storing as C
// Using pointer aritmetic. MatrixSumAllOnGPU implement same logic but using CUDA threads.
void initialData(float *ip, const int size)
{
    int i;

    for(i = 0; i < size; i++)
    {
        ip[i] = (float)( rand() & 0xFF ) / 10.0f;
    }
}

void MatrixSumAll(float *A, float *B, float *C, const int nx, const int ny)
{
    float *ia = A;
    float *ib = B;
    float *ic = C;

    for (int iy = 0; iy < ny; iy++)
    {
        for (int ix = 0; ix < nx; ix++)
        {
            ic[ix] = ia[ix] + ib[ix];
        }

        ia += nx;
        ib += nx;
        ic += nx;
    }

    return;
}

void checkResult(float *hostRef, float *gpuRef, const int N)
{
    double epsilon = 1.0E-8;

    for (int i = 0; i < N; i++)
    {
        if (abs(hostRef[i] - gpuRef[i]) > epsilon)
        {
            printf("host %f gpu %f ", hostRef[i], gpuRef[i]);
            printf("Arrays do not match.\n\n");
            break;
        }
    }
}

// grid 2D block 2D
__global__ void MatrixSumAllOnGPU(float *A, float *B, float *C, int NX, int NY)
{
    unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int idx = iy * NX + ix;

    if (ix < NX && iy < NY)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int nx = 1 << 14;
    int ny = 1 << 14;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // malloc host memory
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initialize data at host side
    size_t iStart = seconds();
    initialData(h_A, nxy);
    initialData(h_B, nxy);
    size_t laps = seconds() - iStart;

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);

    // add matrix at host side for result checks
    iStart = seconds();
    MatrixSumAll (h_A, h_B, hostRef, nx, ny);
    laps = seconds() - iStart;

    // malloc device global memory
    float *d_MatA, *d_MatB, *d_MatC;
    CHECK(hipMalloc((void **)&d_MatA, nBytes));
    CHECK(hipMalloc((void **)&d_MatB, nBytes));
    CHECK(hipMalloc((void **)&d_MatC, nBytes));

    // transfer data from host to device
    CHECK(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice));

    // invoke kernel at host side
    int dimx = 32;
    int dimy = 32;

    if(argc > 2)
    {
        dimx = atoi(argv[1]);
        dimy = atoi(argv[2]);
    }

    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // execute the kernel
    CHECK(hipDeviceSynchronize());
    iStart = seconds();
    MatrixSumAllOnGPU<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
    CHECK(hipDeviceSynchronize());
    laps = seconds() - iStart;
    printf("MatrixSumAllOnGPU (%d,%d), (%d,%d) \n elapsed %d ms\n", int(grid.x), int(grid.y), int(block.x), int(block.y), int(laps));
    CHECK(hipGetLastError());

    // copy kernel result back to host side
    CHECK(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost));

    // check device results
    checkResult(hostRef, gpuRef, nxy);

    // free device global memory
    CHECK(hipFree(d_MatA));
    CHECK(hipFree(d_MatB));
    CHECK(hipFree(d_MatC));

    // free host memory
    free(h_A);
    free(h_B);
    free(hostRef);
    free(gpuRef);

    // reset device
    CHECK(hipDeviceReset());

    return EXIT_SUCCESS;
}
